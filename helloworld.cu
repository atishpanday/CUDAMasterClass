#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void hello() {
	int tid = threadIdx.x;
	if(tid == 0) {
		printf("Hello ");
	}
	else {
		printf("World ");
	}
}

int main () {
	hello<<<1, 2>>>();
	hipDeviceSynchronize();
	return 0;
}
