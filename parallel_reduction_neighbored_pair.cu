#include <hip/hip_runtime.h>


#include <stdio.h>
#include <cstdlib>

__global__ void neighbored_pair_reduction(int* input, int* temp, int size) {
	int tid = threadIdx.x;
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

	if (gid > size) {
		return;
	}

	for (int offset = 1; offset < blockDim.x; offset *= 2) {	// [0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15] -> [1, 5, 9, 13, 17, 21, 25, 29]
		if (tid % (2 * offset) == 0) {
			input[gid] += input[gid + offset];
		}
		__syncthreads();
	}

	if (tid == 0) {
		temp[blockIdx.x] = input[gid];
	}
}

int main() {
	const int N = 1 << 14;
	const int block_size = 128;
	const int num_blocks = N / block_size;
	int arr[N];
	int h_temp[num_blocks];
	int cpu_result = 0;

	for (int i = 0; i < N; i++) {
		arr[i] = rand() * 10 / RAND_MAX;
	}

	for (int i = 0; i < N; i++) {
		cpu_result += arr[i];
	}

	dim3 block(block_size);
	dim3 grid(num_blocks);

	int* d_arr, * d_temp;

	hipMalloc((void**) &d_arr, N*sizeof(int));
	hipMalloc((void**) &d_temp, num_blocks*sizeof(int));

	hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(d_temp, 0, num_blocks * sizeof(int));

	neighbored_pair_reduction<<<grid, block>>>(d_arr, d_temp, N);
	hipDeviceSynchronize();

	hipMemcpy(h_temp, d_temp, num_blocks * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 1; i < num_blocks; i++) {
		h_temp[0] += h_temp[i];
	}

	printf("Number of blocks: %d, CPU results: %d, GPU results : %d", num_blocks, cpu_result, h_temp[0]);

	hipFree(d_arr);
	hipFree(d_temp);
	hipDeviceReset();

	return 0;
}