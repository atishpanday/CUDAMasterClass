#include <hip/hip_runtime.h>


#include <stdio.h>
#include <cstdlib>

__global__ void neighbored_pair_reduction(int* input, int* temp, int size) {
	int tid = threadIdx.x;
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

	if (gid > size) {
		return;
	}
	
	if(blockDim.x >= 512 && tid < 256) {
		input[gid] += input[gid + 256];
	}
	__syncthreads();
	
	if(blockDim.x >= 256 && tid < 128) {
		input[gid] += input[gid + 128];
	}
	__syncthreads();
	
	if(blockDim.x >= 128 && tid < 64) {
		input[gid] += input[gid + 64];
	}
	__syncthreads();
	
	volatile int* i_data = input + blockDim.x * blockIdx.x;
	
	if(tid <= 32) {
		i_data[tid] += i_data[tid + 32];
		i_data[tid] += i_data[tid + 16];
		i_data[tid] += i_data[tid + 8];
		i_data[tid] += i_data[tid + 4];
		i_data[tid] += i_data[tid + 2];
		i_data[tid] += i_data[tid + 1];
	}

	if (tid == 0) {
		temp[blockIdx.x] = input[gid];
	}
	
}

int cpu_sum(int arr[], int N) {
	int sum = 0;
	for (int i = 0; i < N; i++) {
		sum += arr[i];
	}
	return sum;
}

int main() {
	const int N = 1 << 14;
	const int block_size = 256;
	const int num_blocks = N / block_size;
	int arr[N];
	int h_temp[num_blocks];
	int cpu_result = 0;

	for (int i = 0; i < N; i++) {
		arr[i] = (int) 10.0 * (rand() / RAND_MAX + 1.0);
	}

	cpu_result = cpu_sum(arr, N);

	dim3 block(block_size);
	dim3 grid(num_blocks);

	int* d_arr, * d_temp;

	hipMalloc((void**) &d_arr, N*sizeof(int));
	hipMalloc((void**) &d_temp, num_blocks*sizeof(int));

	hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(d_temp, 0, num_blocks * sizeof(int));

	neighbored_pair_reduction<<<grid, block>>>(d_arr, d_temp, N);
	hipDeviceSynchronize();

	hipMemcpy(h_temp, d_temp, num_blocks * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 1; i < num_blocks; i++) {
		h_temp[0] += h_temp[i];
	}

	printf("Number of blocks: %d, CPU results: %d, GPU results : %d", num_blocks, cpu_result, h_temp[0]);

	hipFree(d_arr);
	hipFree(d_temp);
	hipDeviceReset();

	return 0;
}
